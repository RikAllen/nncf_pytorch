#include "hip/hip_runtime.h"
#include "common_cuda_funcs.cuh"
#include "common_cuda_defs.cuh"

#define CUDA_CODE
#include "quantization/dla_sw_model.h"
#include <hip/hip_fp16.h>

enum class ScaleType
{
    SINGLE_SCALE,
    PER_WEIGHT_CHANNEL,
    PER_ACTIVATION_CHANNEL
};


ScaleType get_scale_type(const at::Tensor& input, const at::Tensor& input_low, const at::Tensor& input_range)
{
    TORCH_CHECK(input_low.dim() == input_range.dim(), "input_low and input_range have different dimensionality");
    uint64_t scale_dim = input_range.dim();
    for (int i = 0; i < scale_dim; i++)
    {
        TORCH_CHECK(input_low.size(i) == input_range.size(i), "input_low and input_range have different dimension sizes");
    }

    uint64_t scale_count = input_range.numel();

    if (scale_dim > 0)
    {
        // For (NxCxHxW) input/output tensors, it is assumed that input_range is
        // either (1) for single-scale quantization, or (Nx1x1x1) for
        // per-channel scale weights quantization, or (1xCx1x1) for per-channel
        // activation quantization
        if (input_range.size(0) > 1)
        {
            TORCH_CHECK(input_range.size(0) == input.size(0), "Scale count and weights input channel count is different");
            TORCH_CHECK(input_range.size(0) == scale_count, "Scale shape is not flat");
            return ScaleType::PER_WEIGHT_CHANNEL;
        }
        else if (scale_dim >= 2 and input_range.size(1) > 1)
        {
            TORCH_CHECK(input_range.size(1) == input.size(1), "Scale count and activations channel count is different");
            TORCH_CHECK(input_range.size(1) == scale_count, "Scale shape is not flat");
            return  ScaleType::PER_ACTIVATION_CHANNEL;
        }
    }

    return ScaleType::SINGLE_SCALE;
}


namespace {

template <typename scalar_t>
__device__ void fakeQuantize(
        scalar_t* __restrict__ output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ input_low,
        const scalar_t* __restrict__ input_range,
        const scalar_t levels
        ) {
    scalar_t s = (levels - 1) / (*input_range);
    (*output) = round((min(max((*input), (*input_low)), (*input_low) + (*input_range)) - (*input_low)) * s) / s + (*input_low);
}

template <typename scalar_t>
__global__ void q_cuda_forward_kernel(
        scalar_t* __restrict__ output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ input_low,
        const scalar_t* __restrict__ input_range,
        const scalar_t levels,
        const uint64_t size,
        const uint64_t contiguous_elements_per_scale,
        const uint64_t scale_count) {
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        // "Scales" are derived from input_low/input_range
        uint64_t scale_idx = static_cast<uint64_t>(idx / contiguous_elements_per_scale) % scale_count;
        fakeQuantize<scalar_t>((output + idx), (input + idx), input_low + scale_idx, input_range + scale_idx, levels);
    }
}

template <typename scalar_t>
__device__ void calcGrad(
        scalar_t* __restrict__ val_grad_input,
        scalar_t* __restrict__ val_grad_input_low,
        scalar_t* __restrict__ val_grad_input_range,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ output,
        const scalar_t range_low,
        const scalar_t range_high,
        const scalar_t reverted_range,
        const scalar_t val_low_grad) {
    *val_grad_input_range = 0;
    *val_grad_input_low = 0;
    *val_grad_input = 0;

    if ((*input) < range_low) {
        (*val_grad_input_range) = val_low_grad * (*grad_output);
        (*val_grad_input_low) = (*grad_output);
    } else if ((*input) > range_high) {
        (*val_grad_input_range) = (*grad_output);
        (*val_grad_input_low) = (*grad_output);
    } else {
        (*val_grad_input_range) = (*grad_output) * (((*output) - (*input)) * reverted_range);
        (*val_grad_input) = (*grad_output);
    }
}


template <typename scalar_t>
__global__ void q_single_scale_cuda_backward_kernel(
        scalar_t* __restrict__ grad_input,
        scalar_t* __restrict__ grad_input_low,
        scalar_t* __restrict__ grad_input_range,
        scalar_t* __restrict__ dev_tmp_range,
        scalar_t* __restrict__ dev_tmp_low,
        int32_t* __restrict__ dev_last_block_counter_range,
        int32_t* __restrict__ dev_last_block_counter_low,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ input_low,
        const scalar_t* __restrict__ input_range,
        const scalar_t levels,
        const scalar_t level_low,
        const scalar_t level_high,
        const size_t size) {
    const uint16_t tidx = threadIdx.x;
    const uint32_t bidx = blockIdx.x;
    const uint64_t gtidx = bidx * CUDA_MAX_NUM_THREADS_PER_BLOCK + tidx;
    const uint64_t grid_size = CUDA_MAX_NUM_THREADS_PER_BLOCK * gridDim.x;

    scalar_t sum_range = 0, sum_low = 0;
    scalar_t output, val_grad_input_range, val_grad_input_low;
    scalar_t alpha = level_low / level_high;
    scalar_t range_low = (*input_low);
    scalar_t range_high = (*input_low) + (*input_range);
    scalar_t reverted_range = 1 / (*input_range);
    for (size_t i = gtidx; i < size; i += grid_size) {
        fakeQuantize<scalar_t>(&output, (input + i), input_low, input_range, levels);
        calcGrad<scalar_t>((grad_input + i), &val_grad_input_low, &val_grad_input_range, (grad_output + i),
                 (input + i), &output, range_low, range_high, reverted_range, alpha);
        sum_range += val_grad_input_range;
        sum_low += val_grad_input_low;
    }

    __shared__ scalar_t sh_grad_range[CUDA_MAX_NUM_THREADS_PER_BLOCK];
    __shared__ scalar_t sh_grad_low[CUDA_MAX_NUM_THREADS_PER_BLOCK];
    reduce_with_shared_memory<scalar_t>(sh_grad_range, sum_range, tidx, bidx, dev_tmp_range, dev_last_block_counter_range, grad_input_range, gridDim.x);
    reduce_with_shared_memory<scalar_t>(sh_grad_low, sum_low, tidx, bidx, dev_tmp_low, dev_last_block_counter_low, grad_input_low, gridDim.x);
}



template <typename scalar_t>
__global__ void q_scale_per_weight_channel_cuda_backward_kernel(
        scalar_t* __restrict__ grad_input,
        scalar_t* __restrict__ grad_input_low,
        scalar_t* __restrict__ grad_input_range,
        scalar_t* __restrict__ dev_tmp_range,
        scalar_t* __restrict__ dev_tmp_low,
        int32_t* __restrict__ dev_last_block_counter_range,
        int32_t* __restrict__ dev_last_block_counter_low,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ input_low,
        const scalar_t* __restrict__ input_range,
        const scalar_t levels,
        const scalar_t level_low,
        const scalar_t level_high,
        const size_t elements_per_scale) {
    const uint16_t tidx = threadIdx.x;
    const uint32_t scale_idx = blockIdx.x;
    const uint32_t per_scale_block_idx = blockIdx.y;

    const uint64_t per_scale_tidx = per_scale_block_idx * CUDA_MAX_NUM_THREADS_PER_BLOCK + tidx;
    const uint32_t total_blocks_per_scale = gridDim.y;
    const uint64_t total_threads_per_scale = total_blocks_per_scale * CUDA_MAX_NUM_THREADS_PER_BLOCK;

    // Applying scale data offsets
    input_low += scale_idx;
    input_range += scale_idx;
    dev_tmp_low += scale_idx * total_blocks_per_scale;
    dev_tmp_range += scale_idx * total_blocks_per_scale;
    dev_last_block_counter_low += scale_idx;
    dev_last_block_counter_range += scale_idx;
    grad_input_low += scale_idx;
    grad_input_range += scale_idx;

    const size_t offset_for_scaled_quantized_elements = scale_idx * elements_per_scale;
    input += offset_for_scaled_quantized_elements;
    grad_input += offset_for_scaled_quantized_elements;
    grad_output += offset_for_scaled_quantized_elements;

    scalar_t per_thread_grad_sum_range = 0, per_thread_grad_sum_low = 0;
    scalar_t output, val_grad_input_range, val_grad_input_low;
    scalar_t alpha = level_low / level_high;
    scalar_t range_low = (*input_low);
    scalar_t range_high = (*input_low) + (*input_range);
    scalar_t reverted_range = 1 / (*input_range);
    for (size_t i = per_scale_tidx; i < elements_per_scale; i += total_threads_per_scale) {
        fakeQuantize<scalar_t>(&output, (input + i), input_low, input_range, levels);
        calcGrad<scalar_t>((grad_input + i), &val_grad_input_low, &val_grad_input_range, (grad_output + i),
                 (input + i), &output, range_low, range_high, reverted_range, alpha);
        per_thread_grad_sum_range += val_grad_input_range;
        per_thread_grad_sum_low += val_grad_input_low;
    }

    __shared__ scalar_t sh_grad_range[CUDA_MAX_NUM_THREADS_PER_BLOCK];
    __shared__ scalar_t sh_grad_low[CUDA_MAX_NUM_THREADS_PER_BLOCK];
    reduce_with_shared_memory<scalar_t>(sh_grad_range, per_thread_grad_sum_range, tidx, per_scale_block_idx, dev_tmp_range, dev_last_block_counter_range, grad_input_range, total_blocks_per_scale);
    reduce_with_shared_memory<scalar_t>(sh_grad_low, per_thread_grad_sum_low, tidx, per_scale_block_idx, dev_tmp_low, dev_last_block_counter_low, grad_input_low, total_blocks_per_scale);
}


template <typename scalar_t>
__global__ void q_scale_per_activation_channel_cuda_backward_kernel(
        scalar_t* __restrict__ grad_input,
        scalar_t* __restrict__ grad_input_low,
        scalar_t* __restrict__ grad_input_range,
        scalar_t* __restrict__ dev_tmp_range,
        scalar_t* __restrict__ dev_tmp_low,
        int32_t* __restrict__ dev_last_block_counter_range,
        int32_t* __restrict__ dev_last_block_counter_low,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ input_low,
        const scalar_t* __restrict__ input_range,
        const scalar_t levels,
        const scalar_t level_low,
        const scalar_t level_high,
        const int64_t total_elements_per_scale,
        const int64_t contiguous_elements_per_scale,
        const int64_t scale_count,
        const int64_t leading_channel_offset) {
    const uint16_t tidx = threadIdx.x;
    const uint32_t scale_idx = blockIdx.x;
    const uint32_t per_scale_block_idx = blockIdx.y;

    const uint64_t per_scale_tidx = per_scale_block_idx * CUDA_MAX_NUM_THREADS_PER_BLOCK + tidx;
    const uint32_t total_blocks_per_scale = gridDim.y;
    const uint64_t total_threads_per_scale = total_blocks_per_scale * CUDA_MAX_NUM_THREADS_PER_BLOCK;

    // Applying scale data offsets
    input_low += scale_idx;
    input_range += scale_idx;
    dev_tmp_low += scale_idx * total_blocks_per_scale;
    dev_tmp_range += scale_idx * total_blocks_per_scale;
    dev_last_block_counter_low += scale_idx;
    dev_last_block_counter_range += scale_idx;
    grad_input_low += scale_idx;
    grad_input_range += scale_idx;

    scalar_t per_thread_grad_sum_range = 0, per_thread_grad_sum_low = 0;
    scalar_t output, val_grad_input_range, val_grad_input_low;
    scalar_t alpha = level_low / level_high;
    scalar_t range_low = (*input_low);
    scalar_t range_high = (*input_low) + (*input_range);
    scalar_t reverted_range = 1 / (*input_range);


    // The blocks of values belonging to one and the same scale here are interleaved with a period
    // equal to contiguous_elements_per_scale. Will apply an offset to the beginning of the first
    // block of values belonging to the current scale of the thread block, and then, in the for loop, map
    // a contiguously changing loop iteration index into a value-block-skipping offset calculation pattern.

    const size_t initial_offset = scale_idx * contiguous_elements_per_scale;
    input += initial_offset;
    grad_input += initial_offset;
    grad_output += initial_offset;


    for (uint64_t i = per_scale_tidx; i < total_elements_per_scale; i += total_threads_per_scale) {
        size_t additional_offset = (i / contiguous_elements_per_scale) * leading_channel_offset + (i % contiguous_elements_per_scale);
        fakeQuantize<scalar_t>(&output, (input + additional_offset), input_low, input_range, levels);
        calcGrad<scalar_t>((grad_input + additional_offset), &val_grad_input_low, &val_grad_input_range, (grad_output + additional_offset),
                 (input + additional_offset), &output, range_low, range_high, reverted_range, alpha);
        per_thread_grad_sum_range += val_grad_input_range;
        per_thread_grad_sum_low += val_grad_input_low;
    }

    __shared__ scalar_t sh_grad_range[CUDA_MAX_NUM_THREADS_PER_BLOCK];
    __shared__ scalar_t sh_grad_low[CUDA_MAX_NUM_THREADS_PER_BLOCK];
    reduce_with_shared_memory<scalar_t>(sh_grad_range, per_thread_grad_sum_range, tidx, per_scale_block_idx, dev_tmp_range, dev_last_block_counter_range, grad_input_range, total_blocks_per_scale);
    reduce_with_shared_memory<scalar_t>(sh_grad_low, per_thread_grad_sum_low, tidx, per_scale_block_idx, dev_tmp_low, dev_last_block_counter_low, grad_input_low, total_blocks_per_scale);
}


}

at::Tensor q_cuda_forward(
        at::Tensor input,
        at::Tensor input_low,
        at::Tensor input_range,
        int levels) {
    const auto quantized_elements_count = input.numel();

    ScaleType scale_type = get_scale_type(input, input_low, input_range);

    uint64_t contiguous_elements_per_scale = 0;
    uint64_t scale_count = input_range.numel();
    switch (scale_type)
    {
        case ScaleType::PER_ACTIVATION_CHANNEL:
            // Scale count should be equal to 1-st input tensor dimension
            contiguous_elements_per_scale = quantized_elements_count / (input.size(0) * scale_count);
            break;
        case ScaleType::PER_WEIGHT_CHANNEL:
            // Scale count should be equal to 0-th input tensor dimension
            contiguous_elements_per_scale = quantized_elements_count / scale_count;
            break;
        default:
            contiguous_elements_per_scale = quantized_elements_count;
            break;
    }


    auto output = at::empty_like(input);

    PROFILE(AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "q_cuda_forward", ([&] {
          q_cuda_forward_kernel<scalar_t><<<GET_BLOCKS(quantized_elements_count), CUDA_MAX_NUM_THREADS_PER_BLOCK, 0, at::cuda::getCurrentCUDAStream()>>>(
              output.data_ptr<scalar_t>(),
              input.data_ptr<scalar_t>(),
              input_low.data_ptr<scalar_t>(),
              input_range.data_ptr<scalar_t>(),
              levels,
              quantized_elements_count,
              contiguous_elements_per_scale,
              scale_count);
        }));)

    return output;
}


std::vector<at::Tensor> q_single_scale_cuda_backward(at::Tensor grad_output,
        at::Tensor input,
        at::Tensor input_low,
        at::Tensor input_range,
        int levels,
        int level_low,
        int level_high) {

    const auto size = input.numel();
    auto grad_input = at::empty_like(grad_output);

    auto grad_input_range = at::empty({1}, grad_output.options());
    auto grad_input_low = at::empty({1}, grad_output.options());

    auto grid_size = std::min(GET_BLOCKS(size), CUDA_BLOCKS_PER_GRID_FOR_UNIFORM_ELTWISE);
    auto dev_tmp_range = at::empty({grid_size}, grad_output.options());
    auto dev_tmp_low = at::empty({grid_size}, grad_output.options());
    auto dev_last_block_counter_range = at::zeros({1},  at::device(grad_output.options().device()).dtype(at::kInt));
    auto dev_last_block_counter_low = at::zeros({1},  at::device(grad_output.options().device()).dtype(at::kInt));

    PROFILE(AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "q_single_scale_cuda_backward", ([&] {
      q_single_scale_cuda_backward_kernel<scalar_t><<<grid_size, CUDA_MAX_NUM_THREADS_PER_BLOCK, 0, at::cuda::getCurrentCUDAStream()>>>(
          grad_input.data_ptr<scalar_t>(),
          grad_input_low.data_ptr<scalar_t>(),
          grad_input_range.data_ptr<scalar_t>(),
          dev_tmp_range.data_ptr<scalar_t>(),
          dev_tmp_low.data_ptr<scalar_t>(),
          dev_last_block_counter_range.data_ptr<int32_t>(),
          dev_last_block_counter_low.data_ptr<int32_t>(),
          grad_output.data_ptr<scalar_t>(),
          input.data_ptr<scalar_t>(),
          input_low.data_ptr<scalar_t>(),
          input_range.data_ptr<scalar_t>(),
          levels,
          level_low,
          level_high,
          size);
    }));)

    return {grad_input, grad_input_low, grad_input_range};
}



std::vector<at::Tensor> q_scale_per_weight_channel_cuda_backward(at::Tensor grad_output,
        at::Tensor input,
        at::Tensor input_low,
        at::Tensor input_range,
        int levels,
        int level_low,
        int level_high) {

    const auto scale_count = input_range.size(0);
    const auto elements_per_scale = input.numel() / scale_count;

    auto grad_input = at::empty_like(grad_output);

    auto grad_input_low = at::empty(input_range.sizes(), grad_output.options());
    auto grad_input_range = at::empty(input_range.sizes(), grad_output.options());

    dim3 grid_size = get_2d_grid_size_for_per_channel(scale_count);
    auto dev_tmp_range = at::zeros({grid_size.x, grid_size.y}, grad_output.options());
    auto dev_tmp_low = at::zeros({grid_size.x, grid_size.y}, grad_output.options());
    auto dev_last_block_counter_range = at::zeros({grid_size.x, 1},  at::device(grad_output.options().device()).dtype(at::kInt));
    auto dev_last_block_counter_low = at::zeros({grid_size.x, 1},  at::device(grad_output.options().device()).dtype(at::kInt));

    PROFILE(AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "q_single_scale_cuda_backward", ([&] {
              q_scale_per_weight_channel_cuda_backward_kernel<scalar_t><<<grid_size, CUDA_MAX_NUM_THREADS_PER_BLOCK, 0, at::cuda::getCurrentCUDAStream()>>>(
                  grad_input.data_ptr<scalar_t>(),
                  grad_input_low.data_ptr<scalar_t>(),
                  grad_input_range.data_ptr<scalar_t>(),
                  dev_tmp_range.data_ptr<scalar_t>(),
                  dev_tmp_low.data_ptr<scalar_t>(),
                  dev_last_block_counter_range.data_ptr<int32_t>(),
                  dev_last_block_counter_low.data_ptr<int32_t>(),
                  grad_output.data_ptr<scalar_t>(),
                  input.data_ptr<scalar_t>(),
                  input_low.data_ptr<scalar_t>(),
                  input_range.data_ptr<scalar_t>(),
                  levels,
                  level_low,
                  level_high,
                  elements_per_scale);
            }));
    )
    return {grad_input, grad_input_low, grad_input_range};
}


std::vector<at::Tensor> q_scale_per_activation_channel_cuda_backward(at::Tensor grad_output,
        at::Tensor input,
        at::Tensor input_low,
        at::Tensor input_range,
        int levels,
        int level_low,
        int level_high) {

    const auto scale_count = input_range.size(1);
    const auto total_elements_per_scale = input.numel() / scale_count;
    const auto contiguous_elements_per_scale = input.numel() / (scale_count * input.size(0));
    const auto leading_channel_offset = input.numel() / input.size(0);

    auto grad_input = at::empty_like(grad_output);

    auto grad_input_low = at::empty(input_range.sizes(), grad_output.options());
    auto grad_input_range = at::empty(input_range.sizes(), grad_output.options());

    dim3 grid_size = get_2d_grid_size_for_per_channel(scale_count);
    auto dev_tmp_range = at::zeros({grid_size.x, grid_size.y}, grad_output.options());
    auto dev_tmp_low = at::zeros({grid_size.x, grid_size.y}, grad_output.options());
    auto dev_last_block_counter_range = at::zeros({grid_size.x, 1},  at::device(grad_output.options().device()).dtype(at::kInt));
    auto dev_last_block_counter_low = at::zeros({grid_size.x, 1},  at::device(grad_output.options().device()).dtype(at::kInt));

    PROFILE(
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "q_scale_per_activation_channel_cuda_backward", ([&] {
          q_scale_per_activation_channel_cuda_backward_kernel<scalar_t><<<grid_size, CUDA_MAX_NUM_THREADS_PER_BLOCK, 0, at::cuda::getCurrentCUDAStream()>>>(
              grad_input.data_ptr<scalar_t>(),
              grad_input_low.data_ptr<scalar_t>(),
              grad_input_range.data_ptr<scalar_t>(),
              dev_tmp_range.data_ptr<scalar_t>(),
              dev_tmp_low.data_ptr<scalar_t>(),
              dev_last_block_counter_range.data_ptr<int32_t>(),
              dev_last_block_counter_low.data_ptr<int32_t>(),
              grad_output.data_ptr<scalar_t>(),
              input.data_ptr<scalar_t>(),
              input_low.data_ptr<scalar_t>(),
              input_range.data_ptr<scalar_t>(),
              levels,
              level_low,
              level_high,
              total_elements_per_scale,
              contiguous_elements_per_scale,
              scale_count,
              leading_channel_offset);
        }));
    )
    return {grad_input, grad_input_low, grad_input_range};
}

std::vector<at::Tensor> q_cuda_backward(
        at::Tensor grad_output,
        at::Tensor input,
        at::Tensor input_low,
        at::Tensor input_range,
        int levels,
        int level_low,
        int level_high) {

    ScaleType scale_type = get_scale_type(input, input_low, input_range);

    switch (scale_type)
    {
        case ScaleType::PER_ACTIVATION_CHANNEL:
            return q_scale_per_activation_channel_cuda_backward(
                grad_output,
                input,
                input_low,
                input_range,
                levels,
                level_low,
                level_high);
        case ScaleType::PER_WEIGHT_CHANNEL:
            return q_scale_per_weight_channel_cuda_backward(
                grad_output,
                input,
                input_low,
                input_range,
                levels,
                level_low,
                level_high);
        case ScaleType::SINGLE_SCALE:
        default:
            return q_single_scale_cuda_backward(
                grad_output,
                input,
                input_low,
                input_range,
                levels,
                level_low,
                level_high);
    };
}

/////////////////////////////////////
// BLOCK FLOATING POINT SUPPORT
/////////////////////////////////////
#define BLOCKFP_MAX_BLOCK_SIZE 32

__global__ void block_align_floats_kernel(float* out, float* in, uint32_t exp_width,
    uint32_t mantissa_width, uint32_t block_size,  uint32_t N, uint32_t C, uint32_t HxW, bool sw_rnd) {
 
  int n  = blockIdx.x;
  int hw = blockIdx.y;
  int c  = blockIdx.z;
  int b  = threadIdx.x;

  __shared__ float c_vec[BLOCKFP_MAX_BLOCK_SIZE];
  __shared__ uint32_t max_exp;
  assert (BLOCKFP_MAX_BLOCK_SIZE >= block_size);

  // Load block_size number of values into local array
  int c_idx = c * block_size + b;
  int idx = (n * C + c_idx) * HxW + hw;

  if (c_idx < C) {
    c_vec[b] = in[idx];
  } else {
    // fully initialize c_vec[], as all values participate in max exponent selection
    c_vec[b] = 0.0f;
  }

  // Block gathered c_vec[]
  dla_block_c_vec_cuda (c_vec, &max_exp, b, block_size, exp_width, mantissa_width, sw_rnd, false /* not input layer */);

  // Write blocked c_vec to global out[]
  if (c_idx < C) {
    out[idx] = c_vec[b];
  }
}


/// Performs block align with DLA's folding transform taken into consideration, each instance
/// of this performs block align for a full folded width. 
///
/// The first convolution of a CNN typically has a depth of 3 and non-unity stride and non 1x1 kernels.
/// Given this information, DLA performs folding in which some of the width and height of the tensor is "folded"
/// into the depth of the tensor.
/// e.g. if the input is 3x224x224, stride is 2x2, kernel is 7x7 then DLA's folding will produce:
/// 12x112x112, stride 1x1, kernel 4x4.
/// Therefore blocking needs to take this into account by performing the blocking in a folded manner
/// (while keeping the original tensor unfolded)
/// @param in     Pointer to the original floating point input
/// @param out    Pointer to output buffer
/// @param exp_width      Width of the floating-point exponent (e.g. 5 for half precision, 8 for single precision)
/// @param mantissa_width  Width of the unblocked floating-point mantissa (e.g. 10 for half precision, 23 for single precision, 5 for FP11)
/// @param block_size    The number of elements to be grouped together in a block for a dot product (e.g. 16, 32, etc.)
/// @param N             Batch size (or number of output channels for filters)
/// @param C                Depth or number of channels
/// @param H        Height of the tensor
/// @param W        Width of the tensor
/// @param SY        Stride in the height dimension of the convolution 
/// @param SX        Stride in the width dimension of the convolution
/// @param PY        Padding in the height dimension, currently assumes symmetric padding
/// @param PX        Padding in the width dimension, currently assumes symmetric padding
/// @param sw_rnd      Flag to enable additional rounding only used in software (e.g. subnormal rounding)

__global__ void block_align_folded_inputs_kernel(float *out, float *in, uint32_t exp_width, uint32_t mantissa_width,
      uint32_t block_size, uint32_t N, uint32_t C, uint32_t H, uint32_t W, uint32_t SY, uint32_t SX, uint32_t PY, uint32_t PX, bool sw_rnd) {

  int n = blockIdx.x;
  int w = blockIdx.y;
  int h = blockIdx.z;

  int c = threadIdx.x;
  int sy = threadIdx.y;
  int sx = threadIdx.z;

  // ceil(C * SY * SX / block_size) is how many blocks there are
  // e.g. ceil(3x2x2/16.0) = 1 block per folded height and folded width
  int num_blocks = (int)ceil(C * SY * SX / (float) block_size);
    
  assert (MAX_NUM_BLOCKS >= num_blocks);
  assert (BLOCKFP_MAX_BLOCK_SIZE >= (num_blocks * block_size));

  __shared__ float c_vec[BLOCKFP_MAX_BLOCK_SIZE];
  __shared__ uint32_t max_exp[MAX_NUM_BLOCKS];

  int c_vec_idx = (c * SY + sy) * SX + sx;

  c_vec[c_vec_idx] = 0.0f;

  if (c == 0 && sx == 0 && sy == 0) {
    // initialize the rest to 0 by first thread
    for (int i = C*SY*SX; i < (num_blocks * block_size); ++i) {
      c_vec[i] = 0.0f;
    }
  }


  // Blocking along these loops
  int h_idx = h * SY + sy - PY;
  int w_idx = w * SX + sx - PX;
  int idx = (((n * C) + c) * H + h_idx) * W + w_idx;
  // Bounds check to make sure we're not outside of the tensor
  if ((w_idx < W) &&
    (h_idx < H) &&
    (w_idx >= 0) &&
    (h_idx >= 0)) {

    c_vec[c_vec_idx] = in[idx];
  }


  __syncthreads();


  int idx_within_blk = c_vec_idx % block_size;
  int blk_idx        = c_vec_idx / block_size;
  dla_block_c_vec_cuda (c_vec + blk_idx * block_size, &(max_exp[blk_idx]), idx_within_blk, block_size, exp_width, mantissa_width, sw_rnd, true /*input layer*/);


  if ((w_idx < W) && 
    (h_idx < H) &&
    (w_idx >= 0) &&
    (h_idx >= 0)) {
    out[idx] = c_vec[c_vec_idx];
  }
}


at::Tensor bfp_cuda_forward(
  at::Tensor input,
  uint32_t exp_width, 
  uint32_t mantissa_width, 
  uint32_t block_size,
  uint32_t is_weights) {

  uint32_t N = input.size(0);
  uint32_t C = input.size(1);
  uint32_t HxW = 1;

  for (int d = input.dim()-1; d >=2; d--)
  {
    HxW *= input.size(d);
  }
  // weights are rounded in software, activations are rounded in hardware
  bool sw_rnd = is_weights;
 
  auto output = at::empty_like(input);
  dim3 numBlocks (N, HxW, std::ceil(C/ (float) (block_size)));
  dim3 threadsPerBlock (block_size);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "bfp_cuda_forward", ([&] {
    block_align_floats_kernel<<< numBlocks, threadsPerBlock >>>(
        (float*)output.data_ptr(),
        (float*)input.data_ptr(),
        exp_width, mantissa_width, block_size, 
        N, C, HxW, sw_rnd);
  }));

  return output;
}


at::Tensor bfp_cuda_forward_fold(
    at::Tensor input,
    uint32_t exp_width, 
    uint32_t mantissa_width, 
    uint32_t block_size,
    uint32_t is_weights,
    unsigned int PX,
    unsigned int PY,
    unsigned int SX,
    unsigned int SY) {

    uint32_t N = input.size(0);
    uint32_t C = input.size(1);
    uint32_t W = input.size(2);
    uint32_t H = input.size(3);
    auto output = at::empty_like(input);

    bool sw_rnd = is_weights;
  
    int32_t w_end = std::ceil((W + 2 * PX) / (float) SX);
    int32_t h_end = std::ceil((H + 2 * PY) / (float) SY);

    dim3 numBlocks(N, w_end, h_end);
    dim3 threadsPerBlock (C, SY, SX);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "bfp_cuda_forward_fold", ([&] {
      block_align_folded_inputs_kernel<<<numBlocks, threadsPerBlock>>>(
        (float*)output.data_ptr(), 
        (float*)input.data_ptr(),
        exp_width, mantissa_width, block_size, 
        N, C, H, W, SY, SX, PY, PX, sw_rnd);
    }));

    return output;
}
